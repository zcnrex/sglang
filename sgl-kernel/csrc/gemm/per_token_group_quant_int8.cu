#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <c10/util/Float8_e4m3fn.h>

#include <cmath>
#include <flashinfer/vec_dtypes.cuh>

#include "utils.h"

__device__ __forceinline__ float GroupReduceMax(float val, const int tid) {
  unsigned mask = 0xffff;

  val = fmaxf(val, __shfl_xor_sync(mask, val, 8));
  val = fmaxf(val, __shfl_xor_sync(mask, val, 4));
  val = fmaxf(val, __shfl_xor_sync(mask, val, 2));
  val = fmaxf(val, __shfl_xor_sync(mask, val, 1));
  return val;
}

template <typename T, int GROUPS_PER_BLOCK = 16>
__global__ void per_token_group_quant_int8_kernel(
    const T* __restrict__ input,
    void* __restrict__ output_q,
    float* __restrict__ output_s,
    const int group_size,
    const int num_groups,
    const float eps,
    const float int8_min,
    const float int8_max) {
  const int threads_per_group = 16;
  const int local_group_id = threadIdx.x / threads_per_group;
  const int lane_id = threadIdx.x % threads_per_group;

  const int block_group_id = blockIdx.x * GROUPS_PER_BLOCK;
  const int block_group_offset = (block_group_id + local_group_id) * group_size;

  float local_absmax = eps;

  const T* group_input = input + block_group_offset;
  int8_t* group_output = static_cast<int8_t*>(output_q) + block_group_offset;
  float* scale_output = output_s + (block_group_id + local_group_id);

  constexpr uint32_t vec_size = 16 / sizeof(T);
  using vec_t = flashinfer::vec_t<T, vec_size>;

  const int32_t num_vec_elems = group_size / vec_size;

  for (int32_t i = lane_id; i < num_vec_elems; i += 16) {
    vec_t input_vec;
    input_vec.cast_load(group_input + i * vec_size);

#pragma unroll
    for (uint32_t j = 0; j < vec_size; ++j) {
      float val = static_cast<float>(input_vec[j]);
      float abs_val = fabsf(val);
      local_absmax = fmaxf(local_absmax, abs_val);
    }
  }

  local_absmax = GroupReduceMax(local_absmax, lane_id);

  const float y_s = local_absmax / int8_max;

  if (lane_id == 0) {
    *scale_output = y_s;
  }

  for (int32_t i = lane_id; i < num_vec_elems; i += 16) {
    vec_t input_vec;
    input_vec.cast_load(group_input + i * vec_size);

#pragma unroll
    for (uint32_t j = 0; j < vec_size; ++j) {
      float val = static_cast<float>(input_vec[j]);
      float q_val = fminf(fmaxf(val / y_s, int8_min), int8_max);
#ifdef USE_ROCM
      float dst = std::nearbyint(q_val);
      dst = std::clamp(dst, int8_min, int8_max);
      group_output[i * vec_size + j] = static_cast<int8_t>(dst);
#else
      group_output[i * vec_size + j] = int8_t(q_val);
#endif
    }
  }
}

void sgl_per_token_group_quant_int8(
    torch::Tensor input,
    torch::Tensor output_q,
    torch::Tensor output_s,
    int64_t group_size,
    double eps,
    double int8_min,
    double int8_max) {
  CHECK_INPUT(input);
  CHECK_INPUT(output_q);
  CHECK_INPUT(output_s);

  const int num_groups = input.numel() / group_size;

  CHECK_EQ(input.numel() % group_size, 0);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  constexpr int THREADS_PER_GROUP = 16;

  int groups_per_block = 1;

  if (num_groups % 16 == 0) {
    groups_per_block = 16;
  } else if (num_groups % 8 == 0) {
    groups_per_block = 8;
  } else if (num_groups % 4 == 0) {
    groups_per_block = 4;
  } else if (num_groups % 2 == 0) {
    groups_per_block = 2;
  }

#define LAUNCH_KERNEL(T, GPB)                                                           \
  do {                                                                                  \
    constexpr int GROUPS_PER_BLOCK = GPB;                                               \
    dim3 grid((num_groups + GROUPS_PER_BLOCK - 1) / GROUPS_PER_BLOCK);                  \
    dim3 block(GROUPS_PER_BLOCK* THREADS_PER_GROUP);                                    \
    per_token_group_quant_int8_kernel<T, GROUPS_PER_BLOCK><<<grid, block, 0, stream>>>( \
        static_cast<T*>(input.data_ptr()),                                              \
        output_q.data_ptr(),                                                            \
        static_cast<float*>(output_s.data_ptr()),                                       \
        group_size,                                                                     \
        num_groups,                                                                     \
        (float)eps,                                                                     \
        (float)int8_min,                                                                \
        (float)int8_max);                                                               \
  } while (0)

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FLOAT_FP16(input.scalar_type(), scalar_t, [&] {
    if (groups_per_block == 16) {
      LAUNCH_KERNEL(scalar_t, 16);
    } else if (groups_per_block == 8) {
      LAUNCH_KERNEL(scalar_t, 8);
    } else if (groups_per_block == 4) {
      LAUNCH_KERNEL(scalar_t, 4);
    } else if (groups_per_block == 2) {
      LAUNCH_KERNEL(scalar_t, 2);
    } else {
      LAUNCH_KERNEL(scalar_t, 1);
    }
    return true;
  });

#undef LAUNCH_KERNEL
}
